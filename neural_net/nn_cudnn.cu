#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <random>
#include <iostream>
#include <hipDNN.h>
#include <hipblas.h>
#include <cmath>
#include <algorithm>




#define CHECK_CUDA(call){               \
    hipError_t err = call;             \
    if (err != hipSuccess){            \
        std::cerr<<"CUDA Error in file "<<__FILE__<<" Line number:"<<__LINE__<< "  Error: "<< hipGetErrorString(err) <<std::endl;  \
        exit(EXIT_FAILURE);             \
    }                                   \
}                                       \

#define CHECK_CUDNN(call)                                                         \
  do {                                                                             \
    hipdnnStatus_t status = (call);                                                \
    if (status != HIPDNN_STATUS_SUCCESS) {                                          \
      std::cerr                                                                \
        << "cuDNN Error in file " << __FILE__                                    \
        << " at line " << __LINE__                                               \
        << ": " << hipdnnGetErrorString(status)                                   \
        << std::endl;                                                             \
      std::exit(EXIT_FAILURE);                                                    \
    }                                                                              \
  } while (0)

#define CHECK_CUBLAS(call)                                                    \
  do {                                                                         \
    hipblasStatus_t status = (call);                                            \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                     \
      std::cerr                                                             \
        << "cuBLAS Error in " << __FILE__                                     \
        << " at line " << __LINE__                                            \
        << ": " << status                                                    \
        << std::endl;                                                         \
      std::exit(EXIT_FAILURE);                                                \
    }                                                                          \
  } while (0)



int INPUT_SIZE = 2, HIDDEN_SIZE = 3, OUTPUT_SIZE = 1;
float LR = 0.1f;
int EPOCHS = 10000;


// Choose seed as 42
std::mt19937 gen(42);

// Define the distribution range [-1, +1]
std::uniform_real_distribution<float> dist(-1.0, 1.0);


void random_initialize(float *arr, int size){

    for(int i=0; i<size; i++){
        arr[i] = dist(gen);
    }
}

__global__ void mse_backward_kernel(float *Y, float *Y_true, float *dY, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        dY[i] = 2.0f * (Y[i] - Y_true[i]) / float(N);
    }
}



int main(int argc, char *argv[]){

    float h_w1[INPUT_SIZE * HIDDEN_SIZE], h_w2[HIDDEN_SIZE * OUTPUT_SIZE];
    // float h_b1[HIDDEN_SIZE],              h_b2[OUTPUT_SIZE];
    float h_out[OUTPUT_SIZE];

    // Training data for XOR
    int dataset_size = 4;
    float h_x[dataset_size * INPUT_SIZE]  = {0,0,   0,1,    1,0,    1,1};
    float h_y[dataset_size * OUTPUT_SIZE] = { 0,     1,      1,      0 };

    random_initialize(h_w1, INPUT_SIZE * HIDDEN_SIZE);
    random_initialize(h_w2, HIDDEN_SIZE * OUTPUT_SIZE);


    float *d_x, *d_y;
    float *d_w1, *d_w2, *d_b1, *d_b2, *d_hidden, *d_a_hidden, *d_out, *d_a_out, *d_y_delta, *d_B_delta;


    float *d_dw1, *d_dw2, *d_db1, *d_db2, *d_dh;
    CHECK_CUDA(hipMalloc(&d_dw1, INPUT_SIZE * HIDDEN_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_dw2, HIDDEN_SIZE * OUTPUT_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_db1, HIDDEN_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_db2, OUTPUT_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_dh, HIDDEN_SIZE * sizeof(float)));


    CHECK_CUDA(hipMalloc(&d_x, dataset_size * INPUT_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_y, dataset_size * OUTPUT_SIZE * sizeof(float)));

    CHECK_CUDA(hipMalloc(&d_w1, INPUT_SIZE * HIDDEN_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_w2, HIDDEN_SIZE * OUTPUT_SIZE * sizeof(float)));

    CHECK_CUDA(hipMalloc(&d_b1, HIDDEN_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_b2, OUTPUT_SIZE * sizeof(float)));

    CHECK_CUDA(hipMalloc(&d_hidden, HIDDEN_SIZE * sizeof(float)));
    CHECK_CUDA(hipMemset(d_hidden, 0, HIDDEN_SIZE * sizeof(float)));

    CHECK_CUDA(hipMalloc(&d_a_hidden, HIDDEN_SIZE * sizeof(float)));
    CHECK_CUDA(hipMemset(d_a_hidden, 0, HIDDEN_SIZE * sizeof(float)));

    CHECK_CUDA(hipMalloc(&d_out, OUTPUT_SIZE * sizeof(float)));
    CHECK_CUDA(hipMemset(d_out, 0, OUTPUT_SIZE * sizeof(float)));

    CHECK_CUDA(hipMalloc(&d_y_delta, OUTPUT_SIZE * sizeof(float)));
    CHECK_CUDA(hipMemset(d_y_delta, 0, OUTPUT_SIZE * sizeof(float)));

    CHECK_CUDA(hipMalloc(&d_a_out, OUTPUT_SIZE * sizeof(float)));
    CHECK_CUDA(hipMemset(d_a_out, 0, OUTPUT_SIZE * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_w1, h_w1, INPUT_SIZE * HIDDEN_SIZE * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_w2, h_w2, HIDDEN_SIZE * OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));

    CHECK_CUDA(hipMemset(d_b1, 0, HIDDEN_SIZE * sizeof(float)));
    CHECK_CUDA(hipMemset(d_b2, 0, OUTPUT_SIZE * sizeof(float)));

    CHECK_CUDA(hipMalloc(&d_B_delta, OUTPUT_SIZE*sizeof(float)));


    // Handles
    hipdnnHandle_t cudnn;
    hipblasHandle_t cublas;
    CHECK_CUDNN(hipdnnCreate(&cudnn));
    CHECK_CUBLAS(hipblasCreate(&cublas));


    // Descriptors for Layer 1: Input -> Hidden
    hipdnnTensorDescriptor_t xDesc, hDesc, b1Desc;
    hipdnnFilterDescriptor_t w1Desc;
    hipdnnConvolutionDescriptor_t conv1Desc;
    hipdnnActivationDescriptor_t act1Desc;

    // Input tensor: N=1, C=INPUT_SIZE, H=W=1
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&xDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
                    xDesc,              //tensDesc
                    HIPDNN_TENSOR_NCHW,  // format
                    HIPDNN_DATA_FLOAT,   // datatype
                    1,                  // batch
                    INPUT_SIZE,         // channels = input features
                    1, 1));             // H, W


    // Filter (weights): K=HIDDEN_SIZE, C=INPUT_SIZE, H=W=1
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&w1Desc));
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(
                    w1Desc,
                    HIPDNN_DATA_FLOAT,   // datatype
                    HIPDNN_TENSOR_NCHW,  // format
                    HIDDEN_SIZE,        // K
                    INPUT_SIZE,         // C
                    1, 1));             // H, W


    // Bias: N=1, C=HIDDEN_SIZE, H=W=1
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&b1Desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
                    b1Desc,             //tensDesc
                    HIPDNN_TENSOR_NCHW,  // format
                    HIPDNN_DATA_FLOAT,   // datatype
                    1,                  // batch
                    HIDDEN_SIZE,        // channels = HIDDEN_SIZE
                    1, 1));             // H, W


    // Convolution descriptor: pad=0, stride=1, dilation=1
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&conv1Desc));
    CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(
                    conv1Desc,
                    0,0,                // pad_h, pad_w
                    1,1,                // stride_h, stride_w
                    1,1,                // dilation_h, dilation_w
                    HIPDNN_CROSS_CORRELATION,
                    HIPDNN_DATA_FLOAT));


    // Activation (sigmoid)
    CHECK_CUDNN(hipdnnCreateActivationDescriptor(&act1Desc));
    CHECK_CUDNN(hipdnnSetActivationDescriptor(
                    act1Desc,
                    HIPDNN_ACTIVATION_SIGMOID,
                    HIPDNN_PROPAGATE_NAN,
                    0.0));
    


    // Output (hidden) tensor: N=1, C=HIDDEN_SIZE, H=W=1
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&hDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
        hDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
        1, HIDDEN_SIZE, 1, 1));


    // Descriptors for Layer 2: Hidden -> Output
    hipdnnTensorDescriptor_t hOutDesc, yDesc, b2Desc, d_y_deltaDesc, d_b1_deltaDesc, d_b2_deltaDesc;
    hipdnnFilterDescriptor_t w2Desc;
    hipdnnConvolutionDescriptor_t conv2Desc;
    hipdnnActivationDescriptor_t act2Desc;

    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&d_y_deltaDesc));
    CHECK_CUDNN( hipdnnSetTensor4dDescriptor(
        d_y_deltaDesc,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        1,            // N = batch size
        OUTPUT_SIZE,  // C = number of biases
        1,            // H
        1             // W
    ) );

    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&d_b1_deltaDesc));
    CHECK_CUDNN( hipdnnSetTensor4dDescriptor(
        d_b1_deltaDesc,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        1,            // N = batch size
        OUTPUT_SIZE,  // C = number of biases
        1,            // H
        1             // W
    ) );

    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&d_b2_deltaDesc));
    CHECK_CUDNN( hipdnnSetTensor4dDescriptor(
        d_b2_deltaDesc,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        1,            // N = batch size
        OUTPUT_SIZE,  // C = number of biases
        1,            // H
        1             // W
    ) );
    
    // Hidden input: N=1, C=HIDDEN_SIZE
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&hOutDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
                    hOutDesc,           // tensDesc
                    HIPDNN_TENSOR_NCHW,  // format
                    HIPDNN_DATA_FLOAT,   // datatype
                    1,                  // batch
                    HIDDEN_SIZE,        // channels = HIDDEN_SIZE
                    1, 1));             // H, W


    // Filter (weights): K=OUTPUT_SIZE, C=HIDDEN_SIZE
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&w2Desc));
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(
                    w2Desc,
                    HIPDNN_DATA_FLOAT,   // datatype
                    HIPDNN_TENSOR_NCHW,  // format
                    OUTPUT_SIZE,        // K
                    HIDDEN_SIZE,        // C
                    1, 1));             // H, W

    // Bias: N=1, C=OUTPUT_SIZE
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&b2Desc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
                    b2Desc,             // tensDesc
                    HIPDNN_TENSOR_NCHW,  // format
                    HIPDNN_DATA_FLOAT,   // datatype
                    1,                  // batch
                    OUTPUT_SIZE,        // channels = OUTPUT_SIZE
                    1, 1));             // H, W


    // Convolution descriptor
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&conv2Desc));
    CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(
                    conv2Desc,
                    0,0,                // pad_h, pad_w
                    1,1,                // stride_h, stride_w
                    1,1,                // dilation_h, dilation_w
                    HIPDNN_CROSS_CORRELATION,
                    HIPDNN_DATA_FLOAT));

    // Activation (sigmoid)
    CHECK_CUDNN(hipdnnCreateActivationDescriptor(&act2Desc));
    CHECK_CUDNN(hipdnnSetActivationDescriptor(
                    act2Desc,
                    HIPDNN_ACTIVATION_SIGMOID,
                    HIPDNN_PROPAGATE_NAN,
                    0.0));

    // Output tensor: N=1, C=OUTPUT_SIZE
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&yDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(
                    yDesc,
                    HIPDNN_TENSOR_NCHW,
                    HIPDNN_DATA_FLOAT,
                    1,
                    OUTPUT_SIZE,
                    1, 1));


    //  -------------------------------- FORWARD ALGO --------------------------------
    hipdnnConvolutionFwdAlgo_t fwdAlgo1, fwdAlgo2;

    hipdnnConvolutionFwdAlgoPerf_t perfFwd1[HIPDNN_CONVOLUTION_FWD_ALGO_COUNT];
    int returnedFwdCount1 = 0;
    CHECK_CUDNN(cudnnGetConvolutionForwardAlgorithm_v7(
                    cudnn,          // handle
                    xDesc,          // xDesc    : hipdnnTensorDescriptor_t
                    w1Desc,          // wDesc    : hipdnnFilterDescriptor_t
                    conv1Desc,      // convDesc : hipdnnConvolutionDescriptor_t
                    hDesc,          // yDesc    : hipdnnTensorDescriptor_t
                    HIPDNN_CONVOLUTION_FWD_ALGO_COUNT,         // “how many results do you want?”
                    &returnedFwdCount1,
                    perfFwd1));
    fwdAlgo1 = perfFwd1[0].algo;


    hipdnnConvolutionFwdAlgoPerf_t perfFwd2[HIPDNN_CONVOLUTION_FWD_ALGO_COUNT];
    int returnedFwdCount2 = 0;
    CHECK_CUDNN(cudnnGetConvolutionForwardAlgorithm_v7(
                    cudnn,          // handle
                    hDesc,          // xDesc    : hipdnnTensorDescriptor_t
                    w2Desc,          // wDesc    : hipdnnFilterDescriptor_t
                    conv2Desc,      // convDesc : hipdnnConvolutionDescriptor_t
                    yDesc,          // yDesc    : hipdnnTensorDescriptor_t
                    HIPDNN_CONVOLUTION_FWD_ALGO_COUNT,         // “how many results do you want?”
                    &returnedFwdCount2,
                    perfFwd2));
    fwdAlgo2 = perfFwd2[0].algo;



    // This function returns the amount of GPU memory workspace the user needs to allocate
    // to be able to call hipdnnConvolutionForward() with the specified algorithm. 
    size_t fwdWorkspaceSize1 = 0, fwdWorkspaceSize2 = 0;
    CHECK_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(
                    cudnn,
                    xDesc,
                    w1Desc,
                    conv1Desc,
                    hDesc,
                    fwdAlgo1,
                    &fwdWorkspaceSize1));

    CHECK_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(
                    cudnn,
                    hDesc,
                    w2Desc,
                    conv2Desc,
                    yDesc,
                    fwdAlgo2,
                    &fwdWorkspaceSize2));
    //  -------------------------------- FORWARD ALGO --------------------------------




    //  -------------------------------- BACKWARD ALGO --------------------------------
    // Backward filter algo

    hipdnnConvolutionBwdFilterAlgoPerf_t perfBwdFilter1[HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_COUNT],
                                        perfBwdFilter2[HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_COUNT];
    int returnedBwdFilterCount1 = 0, returnedBwdFilterCount2 = 0;


    hipdnnConvolutionBwdFilterAlgo_t bwdFilterAlgo1, bwdFilterAlgo2;
    CHECK_CUDNN(cudnnGetConvolutionBackwardFilterAlgorithm_v7(
                    cudnn,
                    xDesc,
                    hDesc,
                    conv1Desc,
                    w1Desc,
                    HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_COUNT,
                    &returnedBwdFilterCount1,
                    perfBwdFilter1));
    bwdFilterAlgo1 = perfBwdFilter1[0].algo;

    CHECK_CUDNN(cudnnGetConvolutionBackwardFilterAlgorithm_v7(
                    cudnn,
                    hDesc,
                    yDesc,
                    conv2Desc,
                    w2Desc,
                    HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_COUNT,
                    &returnedBwdFilterCount2,
                    perfBwdFilter2));
    bwdFilterAlgo2 = perfBwdFilter2[0].algo;

    size_t bwdFilterWksz1 = 0, bwdFilterWksz2 = 0;

    CHECK_CUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
                    cudnn,
                    xDesc,
                    hDesc,
                    conv1Desc,
                    w1Desc,
                    bwdFilterAlgo1,
                    &bwdFilterWksz1));

    CHECK_CUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
                    cudnn,
                    hDesc,
                    yDesc,
                    conv2Desc,
                    w2Desc,
                    bwdFilterAlgo2,
                    &bwdFilterWksz2));
                    
                    
    
    // Backward‑data algo
    hipdnnConvolutionBwdDataAlgo_t bwdDataAlgo1, bwdDataAlgo2;
    hipdnnConvolutionBwdDataAlgoPerf_t perfBwdData1[HIPDNN_CONVOLUTION_BWD_DATA_ALGO_TRANSPOSE_GEMM], perfBwdData2[HIPDNN_CONVOLUTION_BWD_DATA_ALGO_TRANSPOSE_GEMM];
    int returnedBwdDataCount1 = 0, returnedBwdDataCount2 = 0;
    CHECK_CUDNN(cudnnGetConvolutionBackwardDataAlgorithm_v7(
                    cudnn,
                    w1Desc,
                    hDesc,
                    conv1Desc,
                    xDesc,
                    HIPDNN_CONVOLUTION_BWD_DATA_ALGO_TRANSPOSE_GEMM,
                    &returnedBwdDataCount1,
                    perfBwdData1));

    bwdDataAlgo1 = perfBwdData1[0].algo;

    CHECK_CUDNN(cudnnGetConvolutionBackwardDataAlgorithm_v7(
                    cudnn,
                    w2Desc,
                    yDesc,
                    conv2Desc,
                    hDesc,
                    HIPDNN_CONVOLUTION_BWD_DATA_ALGO_TRANSPOSE_GEMM,
                    &returnedBwdDataCount2,
                    perfBwdData2));
    bwdDataAlgo2 = perfBwdData2[0].algo;


    size_t bwdDataWksz1 = 0, bwdDataWksz2 = 0;
    CHECK_CUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(
                    cudnn,
                    w1Desc,
                    hDesc,
                    conv1Desc,
                    xDesc,
                    bwdDataAlgo1,
                    &bwdDataWksz1));
    CHECK_CUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(
                    cudnn,
                    w2Desc,
                    yDesc,
                    conv2Desc,
                    hDesc,
                    bwdDataAlgo2,
                    &bwdDataWksz2));
    //  -------------------------------- BACKWARD ALGO --------------------------------


    // size_t workspaceSize*;

    void *workspace;
    int maxWksz = std::max({fwdWorkspaceSize1, fwdWorkspaceSize2, bwdFilterWksz1, bwdFilterWksz2, bwdDataWksz1, bwdDataWksz2});
    hipMalloc(&workspace, maxWksz);



    float alpha = 1.0f, beta = 0.0f;
    int batch_size = 1;
    int N = batch_size * OUTPUT_SIZE;
    int threads = 32, blocks = (N + threads - 1)/threads;
    float loss;

    for (int epoch=0; epoch<EPOCHS; epoch++){
        loss = 0.0f;
        
        for (int sample=0; sample<dataset_size; sample++){
            int prev_x = sample * INPUT_SIZE;
            int prev_y = sample;
        
            // Copy the current sample to device
            CHECK_CUDA(hipMemcpy(d_x, &h_x[prev_x], INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
            CHECK_CUDA(hipMemcpy(d_y, &h_y[prev_y], OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice));
    
            // Forward pass: Input to Hidden Layer
            // FORMULA y = act (alpha1 * conv(x) + alpha2 * z + bias)
            CHECK_CUDNN(cudnnConvolutionBiasActivationForward(
                            cudnn,              // handle
                            &alpha,             // alpha1
                            xDesc,              // xDesc
                            d_x,                // x
                            w1Desc,             // wDesc
                            d_w1,               // w
                            conv1Desc,          // convDesc,
                            fwdAlgo1,           // algo
                            workspace,          // workSpace
                            fwdWorkspaceSize1,  // workSpaceSizeInBytes
                            &beta,              // alpha2
                            hDesc,              // zDesc
                            d_a_hidden,         // z (pre-activation)
                            b1Desc,             // bDesc   
                            d_b1,               // bias
                            act1Desc,           // actDesc
                            hDesc,              // yDesc
                            d_hidden));         // y (post-activation)
    
            // Forward pass: Hidden to Output Layer
            CHECK_CUDNN(cudnnConvolutionBiasActivationForward(
                            cudnn,              // handle
                            &alpha,             // alpha1
                            hDesc,              // xDesc
                            d_hidden,           // x
                            w2Desc,             // wDesc
                            d_w2,               // w
                            conv2Desc,          // convDesc,
                            fwdAlgo2,           // algo
                            workspace,          // workSpace
                            fwdWorkspaceSize2,  // workSpaceSizeInBytes
                            &beta,              // alpha2
                            yDesc,              // zDesc
                            d_a_out,            // z (pre-activation)
                            b2Desc,             // bDesc   
                            d_b2,               // bias
                            act2Desc,           // actDesc
                            yDesc,              // yDesc
                            d_out));            // y (post-activation)
            
            // Calculate loss (MSE) and gradients for output layer
            mse_backward_kernel<<<blocks,threads>>>(d_out, d_y, d_y_delta, N);
            hipDeviceSynchronize();
    
            // Backward pass
            
            // 1. Backward for output layer bias
            CHECK_CUDNN(hipdnnConvolutionBackwardBias(
                            cudnn,
                            &alpha,
                            d_y_deltaDesc,
                            d_y_delta,
                            &beta,
                            b2Desc,
                            d_db2));
            
            // 2. Backward for output layer weights
            CHECK_CUDNN(hipdnnConvolutionBackwardFilter(
                            cudnn,
                            &alpha,
                            hDesc,              // x descriptor
                            d_hidden,           // x (hidden layer activations)
                            d_y_deltaDesc,      // dy descriptor
                            d_y_delta,          // dy (output gradient)
                            conv2Desc,          // convolution descriptor
                            bwdFilterAlgo2,     // algorithm
                            workspace,          // workspace
                            bwdFilterWksz2,     // workspace size
                            &beta,              // beta
                            w2Desc,             // dw descriptor 
                            d_dw2));            // dw (weight gradient)
            
            // 3. Backward for hidden layer (propagate error)
            CHECK_CUDNN(hipdnnConvolutionBackwardData(
                            cudnn,
                            &alpha,
                            w2Desc,             // w descriptor
                            d_w2,               // w (weights)
                            d_y_deltaDesc,      // dy descriptor
                            d_y_delta,          // dy (output gradient)
                            conv2Desc,          // convolution descriptor
                            bwdDataAlgo2,       // algorithm
                            workspace,          // workspace
                            bwdDataWksz2,       // workspace size
                            &beta,              // beta
                            hDesc,              // dx descriptor
                            d_dh));             // dx (hidden gradient)
                            
            // 4. Apply activation backward for hidden layer gradient
            CHECK_CUDNN(hipdnnActivationBackward(
                            cudnn,
                            act1Desc,           // activation descriptor
                            &alpha,             // alpha
                            hDesc,              // y descriptor
                            d_hidden,           // y (activation output)
                            hDesc,              // dy descriptor
                            d_dh,               // dy (gradient from next layer)
                            hDesc,              // x descriptor
                            d_a_hidden,         // x (activation input)
                            &beta,              // beta
                            hDesc,              // dx descriptor
                            d_dh));             // dx (input gradient)
            
            // 5. Backward for hidden layer bias
            CHECK_CUDNN(hipdnnConvolutionBackwardBias(
                            cudnn,
                            &alpha,
                            hDesc,              // dy descriptor
                            d_dh,               // dy (hidden gradient)
                            &beta,              // beta
                            b1Desc,             // db descriptor
                            d_db1));            // db (bias gradient)
            
            // 6. Backward for input-to-hidden weights
            CHECK_CUDNN(hipdnnConvolutionBackwardFilter(
                            cudnn,
                            &alpha,             // alpha
                            xDesc,              // x descriptor
                            d_x,                // x (input)
                            hDesc,              // dy descriptor
                            d_dh,               // dy (hidden gradient)
                            conv1Desc,          // convolution descriptor
                            bwdFilterAlgo1,     // algorithm
                            workspace,          // workspace
                            bwdFilterWksz1,     // workspace size
                            &beta,              // beta
                            w1Desc,             // dw descriptor
                            d_dw1));            // dw (weight gradient)
            
            // Update weights and biases using SGD
            float learning_rate = -LR;  // Negative because we're performing gradient descent
            
            // Update weights using cublas, FORMULA y := α * x + y
            CHECK_CUBLAS(hipblasSaxpy(cublas,                    // handle
                                     INPUT_SIZE * HIDDEN_SIZE,  // length of x and y
                                    &learning_rate,             // α
                                    d_dw1,                      // x: gradient vector
                                    1,                          // stride in x
                                    d_w1,                       // y: weight vector to update
                                    1));                        // // stride in y
            
            CHECK_CUBLAS(hipblasSaxpy(cublas, HIDDEN_SIZE * OUTPUT_SIZE, 
                                    &learning_rate, d_dw2, 1, d_w2, 1));
            
            // Update biases
            CHECK_CUBLAS(hipblasSaxpy(cublas, HIDDEN_SIZE, 
                                    &learning_rate, d_db1, 1, d_b1, 1));
            
            CHECK_CUBLAS(hipblasSaxpy(cublas, OUTPUT_SIZE, 
                                    &learning_rate, d_db2, 1, d_b2, 1));
            
            // Calculate and accumulate loss for monitoring
            float h_pred[OUTPUT_SIZE], h_true[OUTPUT_SIZE];
            CHECK_CUDA(hipMemcpy(h_pred, d_out, OUTPUT_SIZE * sizeof(float), hipMemcpyDeviceToHost));
            CHECK_CUDA(hipMemcpy(h_true, d_y, OUTPUT_SIZE * sizeof(float), hipMemcpyDeviceToHost));
            
            loss += (h_pred[0] - h_true[0]) * (h_pred[0] - h_true[0]);
        }
        
        // Print loss every 1000 epochs
        if (epoch % 1000 == 0) {
            loss /= dataset_size;
            std::cout << "Epoch " << epoch << ", Loss: " << loss << std::endl;
        }
    }
    

    

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_w1);
    hipFree(d_w2);
    hipFree(d_b1);
    hipFree(d_b2);
    hipFree(d_hidden);
    hipFree(d_a_hidden);
    hipFree(d_out);
    hipFree(d_a_out);

    hipFree(d_dw1);
    hipFree(d_dw2);
    hipFree(d_db1);
    hipFree(d_db2);
    hipFree(d_dh);
    hipFree(workspace);

    
    hipdnnDestroyTensorDescriptor(xDesc);
    hipdnnDestroyTensorDescriptor(hDesc);
    hipdnnDestroyTensorDescriptor(yDesc);
    hipdnnDestroyTensorDescriptor(b1Desc);
    hipdnnDestroyTensorDescriptor(b2Desc);
    hipdnnDestroyTensorDescriptor(d_y_deltaDesc);
    hipdnnDestroyTensorDescriptor(d_b1_deltaDesc);
    hipdnnDestroyTensorDescriptor(d_b2_deltaDesc);
    hipdnnDestroyFilterDescriptor(w1Desc);
    hipdnnDestroyFilterDescriptor(w2Desc);
    hipdnnDestroyConvolutionDescriptor(conv1Desc);
    hipdnnDestroyConvolutionDescriptor(conv2Desc);
    hipdnnDestroyActivationDescriptor(act1Desc);
    hipdnnDestroyActivationDescriptor(act2Desc);

    hipdnnDestroy(cudnn);
    hipblasDestroy(cublas);

}